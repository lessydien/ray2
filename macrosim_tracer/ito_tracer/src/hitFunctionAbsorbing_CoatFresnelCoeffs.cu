#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include <optix.h>
#include <optix_math.h>
//#include <commonStructs.h>
//#include "helpers.h"
#include "rayData.h"
#include "MaterialAbsorbing_hit.h"
#include "Coating_FresnelCoeffs_hit.h"

/****************************************************************************/
/*				variable definitions										*/
/****************************************************************************/

rtDeclareVariable(Mat_hitParams, hitParams, attribute hitParams, ); 
rtDeclareVariable(double, t_hit, attribute t_hit, ); 
rtDeclareVariable(int,               max_depth, , );
rtDeclareVariable(float,               min_flux, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(rayStruct, prd, rtPayload, );
//rtDeclareVariable(rtObject,          top_object, , );
//rtDeclareVariable(rtObject,          top_shadower, , );
rtDeclareVariable(int,               geometryID, attribute geometryID , );
rtDeclareVariable(Coating_FresnelCoeffs_ReducedParams, coating_params, , );

/****************************************************************************/
/*				device functions											*/
/****************************************************************************/

__forceinline__ __device__ void absorbingCoatFresnelCoeffs_anyHit_device()
{
  // if we are intersecting the geometry we started from again, we ignore the intersection
//  if (prd.currentGeometryID==geometryID)
//  {
//    rtIgnoreIntersection();
//  }
}

__forceinline__ __device__ void absorbingCoatFresnelCoeffs_closestHit_device( Mat_hitParams hitParams, double t_hit )
{
  rtPrintf("closest hit ID %i \n", geometryID);
  rtPrintf("flux %.20lf \n", prd.flux);
  hitAbsorbing(prd, hitParams, t_hit,geometryID);
  // if coating wants a reflection we keep the ray running and reflect it
  if (hitCoatingFresnelCoeff(prd, hitParams, coating_params))
	  prd.direction=reflect(prd.direction, hitParams.normal);
  if ( (prd.depth>max_depth) || (prd.flux<min_flux) )
	  prd.running=false;	
}

/********************************************************************************/
/*					OptiX programs												*/
/********************************************************************************/

RT_PROGRAM void anyHit()
{
  absorbingCoatFresnelCoeffs_anyHit_device();
}


RT_PROGRAM void closestHit()
{
//  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
//  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

//  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
  absorbingCoatFresnelCoeffs_closestHit_device( hitParams, t_hit );
}
