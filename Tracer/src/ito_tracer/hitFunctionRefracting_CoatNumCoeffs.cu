#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include <optix.h>
#include <optix_math.h>
#include "rayData.h"
#include "rayTracingMath.h"
#include "MaterialRefracting_hit.h"
#include "Coating_NumCoeffs_hit.h"

/****************************************************************************/
/*				variable definitions										*/
/****************************************************************************/

rtDeclareVariable(Mat_hitParams, hitParams, attribute hitParams, ); 
rtDeclareVariable(double, t_hit, attribute t_hit, ); 
rtDeclareVariable(MatRefracting_params, params, , ); 
rtDeclareVariable(int,               max_depth, , );
rtDeclareVariable(float,               min_flux, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float,             scene_epsilon, , );
rtDeclareVariable(rayStruct, prd, rtPayload, );
//rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(int,               geometryID, attribute geometryID , );
rtDeclareVariable(Coating_NumCoeffs_ReducedParams, coating_params, , );

/****************************************************************************/
/*				device functions											*/
/****************************************************************************/

__forceinline__ __device__ void refractingCoatNumCoeffs_anyHit_device()
{
  // this material is opaque, so it fully attenuates all shadow rays
  //prd_shadow.attenuation = make_float3(0);
  //rtTerminateRay();
//  if (prd.currentGeometryID == geometryID)
//  {
//    rtIgnoreIntersection();
//  }

}

__forceinline__ __device__ void refractingCoatNumCoeffs_closestHit_device( Mat_hitParams hitParams, double t_hit )
{
    // see wether coating demands reflection
    bool coat_reflected=hitCoatingNumCoeff(prd, hitParams, coating_params);
    hitRefracting(prd, hitParams, params, t_hit, geometryID, coat_reflected);

  if ( (prd.depth>max_depth) || (prd.flux<min_flux) )
	  prd.running=false;  
}

/********************************************************************************/
/*					OptiX programs												*/
/********************************************************************************/

RT_PROGRAM void anyHit()
{
  refractingCoatNumCoeffs_anyHit_device();
}


RT_PROGRAM void closestHit()
{
  refractingCoatNumCoeffs_closestHit_device( hitParams, t_hit );
}
