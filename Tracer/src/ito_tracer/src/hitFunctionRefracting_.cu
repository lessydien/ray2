#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optix_math_new.h>
#include "hitFunctionRefracting.h"


rtDeclareVariable(double3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(double, t_hit, attribute t_hit, ); 
//rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 


RT_PROGRAM void anyHit()
{
  anyHit_device();
}


RT_PROGRAM void closestHit()
{
//  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
//  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

//  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
  closestHit_device( geometric_normal, t_hit );
}
