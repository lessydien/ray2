#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include <optix.h>
#include <optix_math.h>
#include "rayData.h"
#include "rayTracingMath.h"
#include "MaterialRefracting_hit.h"

/****************************************************************************/
/*				variable definitions										*/
/****************************************************************************/

rtDeclareVariable(Mat_hitParams, hitParams, attribute hitParams, ); 
rtDeclareVariable(double, t_hit, attribute t_hit, ); 
rtDeclareVariable(MatRefracting_params, params, , ); 
rtDeclareVariable(int,               max_depth, , );
rtDeclareVariable(float,               min_flux, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float,             scene_epsilon, , );
rtDeclareVariable(rayStruct, prd, rtPayload, );
//rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(int,               geometryID, attribute geometryID , );

/****************************************************************************/
/*				device functions											*/
/****************************************************************************/

__forceinline__ __device__ void refracting_anyHit_device()
{
  // this material is opaque, so it fully attenuates all shadow rays
  //prd_shadow.attenuation = make_float3(0);
  //rtTerminateRay();
//  if (prd.currentGeometryID == geometryID)
//  {
//    rtIgnoreIntersection();
//  }

}

__forceinline__ __device__ void refracting_closestHit_device( Mat_hitParams hitParams, double t_hit )
{
  if (prd.depth < max_depth)
  {
    rtPrintf("closest hit ID %i \n", geometryID);
	rtPrintf("flux %.20lf \n", prd.flux);
    bool coat_reflected =false;
    hitRefracting(prd, hitParams, params, t_hit, geometryID, coat_reflected);
  }
  else
  {
    rtPrintf("ray stopped in hitRefracting!!");
    prd.running=false; // stop ray
  }
  
}

/********************************************************************************/
/*					OptiX programs												*/
/********************************************************************************/

RT_PROGRAM void anyHit()
{
  refracting_anyHit_device();
}


RT_PROGRAM void closestHit()
{
  refracting_closestHit_device( hitParams, t_hit );
}
