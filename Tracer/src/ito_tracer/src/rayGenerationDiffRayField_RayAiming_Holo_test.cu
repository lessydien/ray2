#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include <optix.h>
#include <optix_math.h>
#include "rayData.h"
#include "randomGenerator.h"
#include "rayTracingMath.h"
#include "DiffRayField_RayAiming_Holo.h"
#include "time.h"

#ifndef PI
	#define PI ((double)3.141592653589793238462643383279502884197169399375105820)
#endif

rtDeclareVariable(DiffRayField_RayAiming_HoloParams, params, , );

//rtDeclareVariable(double3,        params.rayPosStart, , );
//rtDeclareVariable(double3,        params.rayPosEnd, , );
//rtDeclareVariable(double,        params.lambda, , );
//rtDeclareVariable(double,        flux, , );
//rtDeclareVariable(double,        nImmersed, , );
//rtDeclareVariable(unsigned int,        params.width, , );
//rtDeclareVariable(unsigned int,        params.height, , );
//rtDeclareVariable(unsigned int,        params.nrRayDirections, , );
//rtDeclareVariable(rayPosDistrType,        params.posDistrType, , );
//rtDeclareVariable(double3x3,        params.Mrot, , );
//rtDeclareVariable(double3,        params.translation, , );
//rtDeclareVariable(double2,        params.alphaMax, , );
//rtDeclareVariable(double2,        params.alphaMin, , );

//rtDeclareVariable(double3,        params.rayDirection, , );

rtDeclareVariable(long long,        launch_offsetX, , );
rtDeclareVariable(long long,        launch_offsetY, , );

rtDeclareVariable(float,         scene_epsilon, , );
//rtDeclareVariable(float,         diff_epsilon, , );
rtBuffer<diffRayStruct, 1>              output_buffer;
rtBuffer<uint, 1>              seed_buffer;
rtBuffer<double, 2>              holoAngle_buffer;

rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

//#define TIME_VIEW


/**********************************************************/
// device functions for distributing ray positions
/**********************************************************/

__forceinline__ __device__ void init_DiffRay_device(diffRayStruct &prd)
{
	prd.flux=1/(params.epsilon*params.epsilon)*params.flux;
	prd.flux=prd.flux*abs(dot(prd.direction,make_double3(0,0,1)));
	// move ray a short distance out of the caustic
	prd.wavefrontRad=make_double2(-params.epsilon,-params.epsilon);
	prd.mainDirX=make_double3(1,0,0);
	prd.mainDirY=make_double3(0,1,0);
	prd.opl=params.epsilon;
	prd.position=prd.position+params.epsilon*prd.direction;

	// create main directions
	// main directionX is oriented perpendicular to global y-axis, has to be perpendicular to params.rayDirectioncetion and has to be of unit length...
	prd.mainDirX.y=0;
	prd.mainDirY.x=0;
	if (prd.direction.z!=0)
	{
		prd.mainDirX.x=1/sqrt(1-prd.direction.x/prd.direction.z);
		prd.mainDirX.z=-prd.mainDirX.x*prd.direction.x/prd.direction.z;
		prd.mainDirY.y=1/sqrt(1-prd.direction.y/prd.direction.z);
		prd.mainDirY.z=-prd.mainDirY.y*prd.direction.x/prd.direction.z;
	}
	else
	{
		if (prd.direction.x != 0)
		{
			prd.mainDirX.z=1/sqrt(1-prd.direction.z/prd.direction.x);
			prd.mainDirX.x=-prd.mainDirX.z*prd.direction.z/prd.direction.x;
		}
		else
			prd.mainDirX=make_double3(1,0,0);
		if (prd.direction.y != 0)
		{
			prd.mainDirY.z=1/sqrt(1-prd.direction.z/prd.direction.y);
			prd.mainDirY.y=-prd.mainDirY.z*prd.direction.z/prd.direction.y;
		}
		else
			prd.mainDirY=make_double3(0,1,0);
	}	
}

__forceinline__ __device__ void posDistr_RandRect_device(diffRayStruct &prd, diffRayFieldParams &params)
{

	// calc index
	unsigned long long iGes=launch_index+launch_offsetX+launch_offsetY*params.width*params.nrRayDirections.x*params.nrRayDirections.y;

	// calc position indices from 1D index
	unsigned long long iPosX=floorf(iGes/(params.nrRayDirections.x*params.nrRayDirections.y));
	unsigned long long iPosY=floorf(iPosX/params.width);
	iPosX=iPosX % params.width;

	uint32_t x1[5]; // variable for random generator
	unsigned int index=(iPosX+iPosY*params.width) % launch_dim;
	// init random variable
	RandomInit(seed_buffer[index], x1); // rays with same position index use same seed to create their position

	// width of ray field in physical dimension
	double physWidth=params.rayPosEnd.x-params.rayPosStart.x;
	// height of ray field in physical dimension
	double physHeight=params.rayPosEnd.y-params.rayPosStart.y;
	// calc centre of ray field 
	double2 rayFieldCentre=make_double2(params.rayPosStart.x+physWidth/2,params.rayPosStart.y+physHeight/2);

	prd.position.z=0;
	prd.position.x=(Random(x1)-0.5)*physWidth+rayFieldCentre.x;
	prd.position.y=(Random(x1)-0.5)*physHeight+rayFieldCentre.y;
	
	//prd.currentSeed=x1[4]; // don't set current seed as we used the same seed for rays that origin from the same position here...
};

__forceinline__ __device__ void posDistr_GridRect_device(diffRayStruct &prd, diffRayFieldParams &params)
{
	// calc index
	unsigned long long iGes=launch_index+launch_offsetX+launch_offsetY*params.width*params.nrRayDirections.x*params.nrRayDirections.y;

	// calc position indices from 1D index
	unsigned long long iPosX=floorf(iGes/(params.nrRayDirections.x*params.nrRayDirections.y));
	unsigned long long iPosY=floorf(iPosX/params.width);
	iPosX=iPosX % params.width;

	// width of ray field in physical dimension
	double physWidth=params.rayPosEnd.x-params.rayPosStart.x;
	// height of ray field in physical dimension
	double physHeight=params.rayPosEnd.y-params.rayPosStart.y;

	double deltaW=0;
	double deltaH=0;
	// calc increment along x- and y-direction
	if (params.width>0)
		deltaW= (physWidth)/(params.width);
	if (params.height>0)
		// multiple directions per point are listed in y-direction. Therefore the physical height of the rayfield is different from the height of the ray list. This has to be considered here...
		deltaH= (physHeight)/(params.height);
	prd.position.x=params.rayPosStart.x+deltaW/2+iPosX*deltaW;
	prd.position.y=params.rayPosStart.y+deltaH/2+iPosY*deltaH;
	prd.position.z=0;
};

__forceinline__ __device__ void posDistr_RandRad_device(diffRayStruct &prd, diffRayFieldParams &params)
{
	// calc index
	unsigned long long iGes=launch_index+launch_offsetX+launch_offsetY*params.width*params.nrRayDirections.x*params.nrRayDirections.y;

	// calc position indices from 1D index
	unsigned long long iPosX=floorf(iGes/(params.nrRayDirections.x*params.nrRayDirections.y));
	unsigned long long iPosY=floorf(iPosX/params.width);
	iPosX=iPosX % params.width;

	uint32_t x1[5]; // variable for random generator
	unsigned int index=(iPosX+iPosY*params.width) % launch_dim;
	// init random variable
	RandomInit(seed_buffer[index], x1); // rays with same position index use same seed to create their position

	// width of ray field in physical dimension
	double physWidth=params.rayPosEnd.x-params.rayPosStart.x;
	// height of ray field in physical dimension
	double physHeight=params.rayPosEnd.y-params.rayPosStart.y;

	// place a point uniformingly randomly inside the importance area
	double theta=2*PI*Random(x1);
	double r=sqrt(Random(x1));
	double ellipseX=physWidth/2*r*cos(theta);
	double ellipseY=physHeight/2*r*sin(theta);
	double3 exApt=make_double3(1,0,0);
	double3 eyApt=make_double3(0,1,0);
	prd.position=make_double3(0,0,0)+ellipseX*exApt+ellipseY*eyApt;

	//prd.currentSeed=x1[4]; // don't set current seed as we used the same seed for rays that origin from the same position here...
};

__forceinline__ __device__ void posDistr_GridRad_device(diffRayStruct &prd, diffRayFieldParams &params)
{
	// calc index
	unsigned long long iGes=launch_index+launch_offsetX+launch_offsetY*params.width*params.nrRayDirections.x*params.nrRayDirections.y;

	// calc position indices from 1D index
	unsigned long long iPosX=floorf(iGes/(params.nrRayDirections.x*params.nrRayDirections.y));
	unsigned long long iPosY=floorf(iPosX/params.width);
	iPosX=iPosX % params.width;

    // width of ray field in physical dimension
    double physWidth = params.rayPosEnd.x-params.rayPosStart.x;    
    // height of ray field in physical dimension
    double physHeight = params.rayPosEnd.y-params.rayPosStart.y;

	double deltaRx=0;
	double deltaRy=0;
	if (params.width>0)
	{
		deltaRx= (physWidth/2)/double(params.width);
		deltaRy= (physHeight/2)/double(params.width);
	}
	double deltaPhi=0;
	if (params.height>0)
		deltaPhi= (2*PI)/double(params.height);
	// calc r(phi) for given phi and radii of ellipse. see http://en.wikipedia.org/wiki/Ellipse#Polar_form_relative_to_center for reference
	double R=(deltaRx/2+deltaRx*iPosY)*(deltaRy/2+deltaRy*iPosY)/sqrt(pow((deltaRy/2+deltaRy*iPosY)*cos(deltaPhi/2+deltaPhi*iPosX),2)+pow((deltaRx/2+deltaRx*iPosY)*sin((deltaPhi/2+deltaPhi*iPosX)),2));
	// now calc rectangular coordinates from polar coordinates
	prd.position.z=0;
	prd.position.x=cos(deltaPhi/2+deltaPhi*iPosX)*R;
	prd.position.y=sin(deltaPhi/2+deltaPhi*iPosX)*R;
	
};

/**********************************************************/
// device functions for distributing ray directions
/**********************************************************/

__forceinline__ __device__ void dirDistr_Rand_device(diffRayStruct &prd, diffRayFieldParams &params)
{
	uint32_t x1[5]; // variable for random generator			

	RandomInit(prd.currentSeed, x1); // init random variable

	double3 rayAngleCentre=make_double3((params.alphaMax.x+params.alphaMin.x)/2,(params.alphaMax.y+params.alphaMin.y)/2,0);
	double2 rayAngleHalfWidth=make_double2((params.alphaMax.x-params.alphaMin.x)/2,(params.alphaMax.y-params.alphaMin.y)/2);
	// create random angles inside the given range
	double2 phi=make_double2(2*(Random(x1)-0.5)*rayAngleHalfWidth.x+rayAngleCentre.x,2*(Random(x1)-0.5)*rayAngleHalfWidth.y+rayAngleCentre.y);
	// create unit vector with the given angles
	prd.direction=createObliqueVec(phi);//normalize(make_double3(tan(phi.y),tan(phi.x),1));
	// transform raydirection into global coordinate system
	prd.direction=params.Mrot*prd.direction;
	
				
	// create points inside importance area to randomly distribute ray direction
//	double3 rayAngleCentre=make_double3((params.alphaMax.x+params.alphaMin.x)/2,(params.alphaMax.y+params.alphaMin.y)/2,0);
//	double2 impAreaHalfWidth;
//	impAreaHalfWidth.x=(tan(params.alphaMax.x)-tan(params.alphaMin.x))/2;
//	impAreaHalfWidth.y=(tan(params.alphaMax.y)-tan(params.alphaMin.y))/2;
//	double3 dirImpAreaCentre=make_double3(0,0,1);
//	rotateRay(&dirImpAreaCentre, rayAngleCentre);
	// the centre of the importance area is the root of the current geometry + the direction to the imp area centre normalized such that the importance area is 1mm away from the current geometry
//	double3 impAreaRoot=make_double3(0,0,0)+dirImpAreaCentre/dot(make_double3(0,0,1), dirImpAreaCentre);
	// now distribute points inside importance area
//	double theta=2*PI*Random(x1);
//	double r=sqrt(Random(x1));
//	double impAreaX=impAreaHalfWidth.x*r*cos(theta);
//	double impAreaY=impAreaHalfWidth.y*r*sin(theta);
//	double3 tmpPos=impAreaRoot+impAreaX*make_double3(1,0,0)+impAreaY*make_double3(0,1,0);
//	prd.direction=normalize(tmpPos-make_double3(0,0,0));
	// transform raydirection into global coordinate system
//	prd.direction=params.Mrot*prd.direction;

	// save seed for next randomization
	prd.currentSeed=x1[4];
};

__forceinline__ __device__ void dirDistr_RandImpArea_device(diffRayStruct &prd, diffRayFieldParams &params)
{
	uint32_t x1[5]; // variable for random generator			
	
	// declar variables for randomly distributing ray directions via an importance area
	double3 dirImpAreaCentre, tmpPos, impAreaRoot;
	
	RandomInit(prd.currentSeed, x1); // init random variable

	double impAreaX;
	double impAreaY;
			
	// now distribute points inside importance area

	if (params.importanceAreaApertureType==AT_RECT)
	{
		// place temporal point uniformingly randomly inside the importance area
		impAreaX=(Random(x1)-0.5)*2*params.importanceAreaHalfWidth.x;
		impAreaY=(Random(x1)-0.5)*2*params.importanceAreaHalfWidth.y; 
	}
	else 
	{
		if (params.importanceAreaApertureType==AT_ELLIPT)
		{
			double theta=2*PI*Random(x1);
			double r=sqrt(Random(x1));
			impAreaX=params.importanceAreaHalfWidth.x*r*cos(theta);
			impAreaY=params.importanceAreaHalfWidth.y*r*sin(theta);
		}
	}
		
	
	double3 impAreaAxisX=make_double3(1,0,0);
	double3 impAreaAxisY=make_double3(0,1,0);
		
	rotateRay(&impAreaAxisX,params.importanceAreaTilt);
	rotateRay(&impAreaAxisY,params.importanceAreaTilt);

	tmpPos=params.importanceAreaRoot+impAreaX*impAreaAxisX+impAreaY*impAreaAxisY;
	prd.direction=normalize(tmpPos-prd.position);
	// save seed for next randomization
	prd.currentSeed=x1[4];
};

__forceinline__ __device__ void dirDistr_Uniform_device(diffRayStruct &prd, diffRayFieldParams &params)
{
	prd.direction=params.rayDirection;
};

__forceinline__ __device__ void dirDistr_GridRect_device(diffRayStruct &prd, diffRayFieldParams &params)
{
	// calc index
	unsigned long long iGes=launch_index+launch_offsetX+launch_offsetY*params.width*params.nrRayDirections.x*params.nrRayDirections.y;

	// calc position indices from 1D index
	unsigned long long iPosX=floorf(iGes/(params.nrRayDirections.x*params.nrRayDirections.y));
	unsigned long long iPosY=floorf(iPosX/params.width);
	iPosX=iPosX % params.width;

	// calc direction indices from 1D index
	unsigned long long iDirX=(iGes-iPosX*params.nrRayDirections.x*params.nrRayDirections.y-iPosY*params.nrRayDirections.x*params.nrRayDirections.y*params.width) % params.nrRayDirections.x;
	unsigned long long iDirY=floorf((iGes-iPosX*params.nrRayDirections.x*params.nrRayDirections.y-iPosY*params.nrRayDirections.x*params.nrRayDirections.y*params.width)/params.nrRayDirections.x);

    double r; // variable for generating random variables inside an ellipse
	// declar variables for randomly distributing ray directions via an importance area
	double2 impAreaHalfWidth;
	double3 dirImpAreaCentre, tmpPos, impAreaRoot, impAreaAxisX, impAreaAxisY;
	double impAreaX, impAreaY, theta;
	// increment of temporary raypos in x and y 
	double deltaW=0;
	double deltaH=0;

	// calc increment along x- and y-direction
	if (params.nrRayDirections.x>0)
		deltaW= (2*params.importanceAreaHalfWidth.x)/(params.nrRayDirections.x);
	if (params.nrRayDirections.y>0)
		deltaH= (2*params.importanceAreaHalfWidth.y)/(params.nrRayDirections.y);
	impAreaX=-params.importanceAreaHalfWidth.x+deltaW/2+iDirX*deltaW; 
	impAreaY=-params.importanceAreaHalfWidth.y+deltaH/2+iDirY*deltaH; 
	impAreaAxisX=make_double3(1,0,0);
	impAreaAxisY=make_double3(0,1,0);
	rotateRay(&impAreaAxisX,params.importanceAreaTilt);
	rotateRay(&impAreaAxisY,params.importanceAreaTilt);

	tmpPos=params.importanceAreaRoot+impAreaX*impAreaAxisX+impAreaY*impAreaAxisY;
	prd.direction=normalize(tmpPos-prd.position);
};

__forceinline__ __device__ void dirDistr_GridRad_device(diffRayStruct &prd, diffRayFieldParams &params)
{
	// calc index
	unsigned long long iGes=launch_index+launch_offsetX+launch_offsetY*params.width*params.nrRayDirections.x*params.nrRayDirections.y;

	// calc position indices from 1D index
	unsigned long long iPosX=floorf(iGes/(params.nrRayDirections.x*params.nrRayDirections.y));
	unsigned long long iPosY=floorf(iPosX/params.width);
	iPosX=iPosX % params.width;

	// calc direction indices from 1D index
	unsigned long long iDirX=(iGes-iPosX*params.nrRayDirections.x*params.nrRayDirections.y-iPosY*params.nrRayDirections.x*params.nrRayDirections.y*params.width) % params.nrRayDirections.x;
	unsigned long long iDirY=floorf((iGes-iPosX*params.nrRayDirections.x*params.nrRayDirections.y-iPosY*params.nrRayDirections.x*params.nrRayDirections.y*params.width)/params.nrRayDirections.x);

    double r; // variable for generating random variables inside an ellipse
	// declar variables for randomly distributing ray directions via an importance area
	double2 impAreaHalfWidth;
	double3 dirImpAreaCentre, tmpPos, impAreaRoot, impAreaAxisX, impAreaAxisY;
	double impAreaX, impAreaY, theta, deltaPhi;
	// increment of temporary raypos in x and y 
	double deltaRx=0;
	double deltaRy=0;
	
	// calc increment along radial and angular direction
	if (params.nrRayDirections.x>0)
	{
		deltaRx= (params.importanceAreaHalfWidth.x)/double(params.nrRayDirections.x);
		deltaRy= (params.importanceAreaHalfWidth.y)/double(params.nrRayDirections.x);
	}
	if (params.nrRayDirections.y>0)
		deltaPhi= (2*PI)/params.nrRayDirections.y;
	// calc r(phi) for given phi and radii of ellipse. see http://en.wikipedia.org/wiki/Ellipse#Polar_form_relative_to_center for reference
	double R=(deltaRx/2+deltaRx*iDirY)*(deltaRy/2+deltaRy*iDirY)/sqrt(pow((deltaRy/2+deltaRy*iDirY)*cos(deltaPhi/2+deltaPhi*iDirX),2)+pow((deltaRx/2+deltaRx*iDirY)*sin(deltaPhi/2+deltaPhi*iDirX),2));
	if (deltaRy==0)
		R=0;
	// now calc rectangular coordinates from polar coordinates
	impAreaX=cos(deltaPhi*iDirX)*R;
	impAreaY=sin(deltaPhi*iDirX)*R;
	
	impAreaAxisX=make_double3(1,0,0);
	impAreaAxisY=make_double3(0,1,0);
	rotateRay(&impAreaAxisX,params.importanceAreaTilt);
	rotateRay(&impAreaAxisY,params.importanceAreaTilt);

	tmpPos=params.importanceAreaRoot+impAreaX*impAreaAxisX+impAreaY*impAreaAxisY;
	prd.direction=normalize(tmpPos-prd.position);
};


RT_PROGRAM void rayGeneration()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

			switch (params.posDistrType)
			{
				case RAYPOS_GRID_RECT:
					posDistr_GridRect_device(prd, params);
					break;
				case RAYPOS_RAND_RECT:	
					posDistr_RandRect_device(prd, params);
					break;
				case RAYPOS_GRID_RAD:
					posDistr_GridRad_device(prd, params);
					break;
				case RAYPOS_RAND_RAD:
					posDistr_RandRad_device(prd, params);
					break;
				default:
					rtPrintf("RAYPOS_DEFAULT");
					prd.position=make_double3(0,0,0);
					// report error
					break;
			}
			// transform rayposition into global coordinate system
			prd.position=params.Mrot*prd.position+params.translation;

			switch (params.dirDistrType)
			{
				case RAYDIR_UNIFORM:
					dirDistr_Uniform_device(prd, params);
					break;
				case RAYDIR_RAND_RECT:
				    dirDistr_Rand_device(prd, params);
					break;
				case RAYDIR_GRID_RECT:
					dirDistr_GridRect_device(prd, params);
					break;
				case RAYDIR_GRID_RAD:
					dirDistr_GridRad_device(prd, params);
					break;
				default:
					prd.direction=make_double3(0,0,0);
					// report error
					break;
			}

  init_DiffRay_device(prd);  

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

/*****************************************************************
/   DirRandImpArea
/*****************************************************************/

RT_PROGRAM void rayGeneration_DirRandImpArea_PosRandRad()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_RandRad_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_RandImpArea_device(prd, params);
	
	init_DiffRay_device(prd);


  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

RT_PROGRAM void rayGeneration_DirRandImpArea_PosGridRad()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_GridRad_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_RandImpArea_device(prd, params);
	
	init_DiffRay_device(prd);
	
  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}


RT_PROGRAM void rayGeneration_DirRandImpArea_PosRandRect()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif
  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;
  
  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_RandRect_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_RandImpArea_device(prd, params);
	
	init_DiffRay_device(prd);

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 
  

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       break;
    }
  }
  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

RT_PROGRAM void rayGeneration_DirRandImpArea_PosGridRect()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_GridRect_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_RandImpArea_device(prd, params);
	
	init_DiffRay_device(prd);

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }
  //prd.position=make_double3(0,0,100);
  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

/**********************************************************************
/   DirRand
/**********************************************************************/

RT_PROGRAM void rayGeneration_DirRand_PosRandRad()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_RandRad_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_Rand_device(prd, params);
	
	init_DiffRay_device(prd);


  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

RT_PROGRAM void rayGeneration_DirRand_PosGridRad()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_GridRad_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_Rand_device(prd, params);
	
	init_DiffRay_device(prd);
	
  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}


RT_PROGRAM void rayGeneration_DirRand_PosRandRect()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif
  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;
  
  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_RandRect_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_Rand_device(prd, params);
	
	init_DiffRay_device(prd);

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 
  

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       break;
    }
  }
  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

RT_PROGRAM void rayGeneration_DirRand_PosGridRect()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_GridRect_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_Rand_device(prd, params);
	
	//init_DiffRay_device(prd);

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }
  //prd.position=make_double3(0,0,100);
  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

/********************************************************************************************/
//                 Dir_GridRad
/********************************************************************************************/

RT_PROGRAM void rayGeneration_DirGridRad_PosRandRad()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_RandRad_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_GridRad_device(prd, params);
	
	init_DiffRay_device(prd);


  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

RT_PROGRAM void rayGeneration_DirGridRad_PosGridRad()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_GridRad_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_GridRad_device(prd, params);
	
	init_DiffRay_device(prd);
	
  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}


RT_PROGRAM void rayGeneration_DirGridRad_PosRandRect()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif
  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;
  
  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_RandRect_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_GridRad_device(prd, params);
	
	init_DiffRay_device(prd);

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 
  

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       break;
    }
  }
  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

RT_PROGRAM void rayGeneration_DirGridRad_PosGridRect()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_GridRect_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_GridRad_device(prd, params);
	
	init_DiffRay_device(prd);

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }
  //prd.position=make_double3(0,0,100);
  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

/**********************************************************************************************/
//                    DirGridRect
/**********************************************************************************************/

RT_PROGRAM void rayGeneration_DirGridRect_PosRandRad()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_RandRad_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_GridRect_device(prd, params);
	
	init_DiffRay_device(prd);


  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

RT_PROGRAM void rayGeneration_DirGridRect_PosGridRad()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_GridRad_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_GridRect_device(prd, params);
	
	init_DiffRay_device(prd);
	
  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       //prd.result += prd.radiance * prd.attenuation;
       break;
    }
  }

  output_buffer[launch_index] = prd;//.direction;//prd.position;
}


RT_PROGRAM void rayGeneration_DirGridRect_PosRandRect()
{
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif
  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;
  
  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_RandRect_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	dirDistr_GridRect_device(prd, params);
	
	init_DiffRay_device(prd);

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;
  prd.opl=0; 
  

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 
 
  for(;;) 
  {
    rtTrace(top_object, ray, prd);
    if(!prd.running) 
    {
       break;
    }
  }
  output_buffer[launch_index] = prd;//.direction;//prd.position;
}

RT_PROGRAM void rayGeneration_DirUniform_PosGridRect()
{
	rtPrintf("hello \n");
#ifdef TIME_VIEW
  clock_t t0, t1;
  double time;
#endif

  float3 ray_origin = make_float3(0.0f);
  float3 ray_direction = make_float3(0.0f);
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
  
  diffRayStruct prd;
  prd.flux = params.flux;
  prd.depth = 0;

  // set seed for each ray
  prd.currentSeed=seed_buffer[launch_index];

#ifdef TIME_VIEW
  t0=clock();
#endif 

	// calc ray position
	posDistr_GridRect_device(prd, params);	
	// transform rayposition into global coordinate system
	prd.position=params.Mrot*prd.position+params.translation;
	// calc ray direction
	//dirDistr_GridRect_device(prd, params);
	prd.direction=normalize(params.oDetParams.root-prd.position);
	
	init_DiffRay_device(prd);

	double l_phase;
	// add phase according to holoAngle_buffer
	nearestNeighbour_hostdevice(-WIDTH_HOLO_BUFFER/2*DELTA_X_HOLO+DELTA_X_HOLO/2,-HEIGHT_HOLO_BUFFER/2*DELTA_Y_HOLO+DELTA_Y_HOLO/2,DELTA_X_HOLO,DELTA_Y_HOLO, &holoAngle_buffer[make_uint2(0,0)], WIDTH_HOLO_BUFFER,HEIGHT_HOLO_BUFFER, prd.position.x, prd.position.y, &l_phase);

	prd.opl=l_phase;

  prd.currentGeometryID = 0;
  prd.lambda=params.lambda;
  prd.nImmersed=params.nImmersed;
  prd.running=true;

#ifdef TIME_VIEW
  t1 = clock(); 
  time = (double)(t1-t0);
  rtPrintf("time elapsed while creating ray: %lf ms\n", time);
#endif 

  // calc position target position where we want to hit the detector....
	// calc index
	unsigned long long iGes=launch_index+launch_offsetX+launch_offsetY*params.width*params.nrRayDirections.x*params.nrRayDirections.y;

	// calc position indices from 1D index
	unsigned long long iPosX=floorf(iGes/(params.nrRayDirections.x*params.nrRayDirections.y));
	unsigned long long iPosY=floorf(iPosX/params.width);
	iPosX=iPosX % params.width;

	// calc direction indices from 1D index
	unsigned long long iDirX=(iGes-iPosX*params.nrRayDirections.x*params.nrRayDirections.y-iPosY*params.nrRayDirections.x*params.nrRayDirections.y*params.width) % params.nrRayDirections.x;
	unsigned long long iDirY=floorf((iGes-iPosX*params.nrRayDirections.x*params.nrRayDirections.y-iPosY*params.nrRayDirections.x*params.nrRayDirections.y*params.width)/params.nrRayDirections.x);

	double deltaW=(2*params.oDetParams.apertureHalfWidth.x)/params.oDetParams.detPixel.x;
	double deltaH=(2*params.oDetParams.apertureHalfWidth.y)/params.oDetParams.detPixel.y;

	double detX=params.oDetParams.apertureHalfWidth.x-deltaW/2-iDirY*deltaW;
	double detY=params.oDetParams.apertureHalfWidth.y-deltaH/2-iDirX*deltaH;

	double3 detAxisX=make_double3(1,0,0);
	double3 detAxisY=make_double3(0,1,0);
	rotateRay(&detAxisX,params.oDetParams.tilt);
	rotateRay(&detAxisY,params.oDetParams.tilt);

	double3 targetHitPos=params.oDetParams.root+detX*detAxisX+detY*detAxisY;

	// define variables for the ray aiming loop
	bool firstRun=true;

	// init old ray
	diffRayStruct oldRay=prd;
	// move ray back into caustic
	double3 dirOld=prd.direction;
	// init derivative of direction of starting ray with respect to change in position of hit point
	// use a small value to be safe in case the sign is wron in the beginning...
	double3 dDir_dPos=make_double3(0.001,0.001,0.001); 
	double3 diffPos=make_double3(0,0,0);
	double3 hitPosOld=make_double3(0,0,0);

	//unsigned int index=0;
	//// do ray aiming
	//do
	//{
	//	index++;
	//	if (index>50)
	//	{
	//		// some error mechanism
	//		//std::cout << "error in DiffRayField_RayAiming_Holo.traceScene(): ray aiming loop canceled after " << index << " iterations for ray " << jx << "...\n";
	//		break;
	//	}
	//	// init ray with data from last trace
	//	prd=oldRay;
	//	// move ray back into caustic
	//	prd.position=oldRay.position-params.epsilon*oldRay.direction;
	//	// update ray direction
	//	prd.direction=normalize(prd.direction+diffPos*dDir_dPos);
	//	// move ray out of caustic
	//	prd.position=prd.position+params.epsilon*prd.direction;
	//	// create main directions according to new direction
	//	// calc angles with respect to global x- and y-axis
	//	double2 phi=calcAnglesFromVector(prd.direction,params.tilt);
	//	prd.mainDirX=createObliqueVec(make_double2(phi.x,phi.y+M_PI/2));
	//	prd.mainDirY=createObliqueVec(make_double2(phi.x+M_PI/2,phi.y));

	//	// save raydata for next iteration
	//	oldRay=prd;

 // for(;;) 
 // {
 //   //optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, scene_epsilon, RT_DEFAULT_MAX);
 //   rtTrace(top_object, ray, prd);
 //   if(!prd.running) 
 //   {
 //      break;
 //   }
 // }
	//	// calc change in hit position
	//	double3 deltaPos=prd.position-hitPosOld;
	//	// save current position
	//	// in the first run we don't have an old direction and therefore we can not calc dDir_dPos
	//	if (!firstRun)
	//	{
	//		dDir_dPos=(oldRay.direction-dirOld)/deltaPos;
	//		// save starting direction of this iteration
	//		dirOld=oldRay.direction;

	//		// check for division by zero
	//		if(abs(deltaPos.x)<0.0000001)
	//			dDir_dPos.x=0;
	//		if(abs(deltaPos.y)<0.0000001)
	//			dDir_dPos.y=0;
	//		if(abs(deltaPos.z)<0.0000001)
	//			dDir_dPos.z=0;
	//	}
	//	// save current hit pos 
	//	hitPosOld=prd.position;
	//	// calc difference of current hit pos to target hit pos
	//	diffPos=targetHitPos-prd.position;
	//	firstRun=false;

	//}
	//// now check wether we hit it, i.e. wether we are inside the pixel we were aiming at.
	//while ( (abs(dot(diffPos,detAxisX)) > deltaW/2) || (abs(dot(diffPos,detAxisY)) > deltaH/2) );
  //prd.position=make_double3(0,0,100);
  output_buffer[launch_index] = prd;//.direction;//prd.position;
}


RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d)\n", code, (launch_index+launch_offsetX) );
//  output_buffer[launch_index] = prd.position;
}
