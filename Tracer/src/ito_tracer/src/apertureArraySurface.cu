#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/
#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>
#include "rayData.h"
#include "apertureArraySurface_Intersect.h"

rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
//rtDeclareVariable(simMode, mode, , );
rtDeclareVariable(rayStruct, prd, rtPayload, ); // get per-ray-data structure
rtDeclareVariable(ApertureArraySurface_ReducedParams, params, , ); // centre of spherical surface
//rtDeclareVariable(int, materialListLength, , ); 
// variables that are communicated to hit program vie attribute mechanism
rtDeclareVariable(Mat_hitParams, hitParams, attribute hitParams, );
rtDeclareVariable(double, t_hit, attribute t_hit, ); 
rtDeclareVariable(int,               geometryID, attribute geometryID , );

__forceinline__ __device__ Mat_hitParams calcHitParams(double t)
{
  return calcHitParamsApertureArraySurface(prd.position+t*prd.direction, params);
}

RT_PROGRAM void intersect(int)
{ 

	double t=intersectRayApertureArraySurface(prd.position,prd.direction,params);
	// check wether intrersection is within valid interval of t
	if( rtPotentialIntersection( (float)t ) ) 
	{
		//rtPrintf("normal %.20lf %.20lf %.20lf \n", params.orientation.x, params.orientation.y, params.orientation.z);
			
		// calc normal in intersection
		hitParams=calcHitParams(t);
		// save hit paramter
		t_hit=t;
		// pass geometryID to hit-program
		geometryID=params.geometryID;
		// call any hit function of the respective material
		rtReportIntersection( 0 );
	}
}

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(boxmin, boxmax);
}
