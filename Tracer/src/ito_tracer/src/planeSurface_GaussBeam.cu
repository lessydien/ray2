#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>
#include "rayData.h"
#include "PlaneSurface_Intersect.h"

rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(gaussBeamRayStruct, prd, rtPayload, ); // get per-ray-data structure
rtDeclareVariable(PlaneSurface_ReducedParams, params, , ); // normal vector to surface. i.e. part of the definition of the plane surface geometry
// variables that are communicate to the hit program via the attribute mechanism
rtDeclareVariable(gaussBeam_geometricNormal, geometric_normal, attribute geometric_normal, ); // normal to the geometry at the hit-point. at a plane surface this will simply be the normal of the definition of the plane surface
rtDeclareVariable(gaussBeam_t, t_hit, attribute t_hit, );
rtDeclareVariable(int,               geometryID, attribute geometryID , );

/* calc normal to surface at intersection point */
__device__ gaussBeam_geometricNormal calcHitParams(gaussBeam_t t)
{
  gaussBeam_geometricNormal normal;
  normal.normal_baseRay=params.normal;
  normal.normal_waistRayX=params.normal;
  normal.normal_waistRayY=params.normal;
  normal.normal_divRayX=params.normal;
  normal.normal_divRayY=params.normal;
  return normal;
}

/* calc intersection of ray with geometry */
RT_PROGRAM void intersect(int)
{
  gaussBeam_t t;
  // matlab code
  //t=-(ray.xyz-plane.a)'*plane.nNorm/(ray.ek'*plane.nNorm);

  // intersect the centre ray of the gaussian beam with the surface
  t.t_baseRay=intersectRayPlaneSurface(prd.baseRay.position, prd.baseRay.direction, params);
  // check wether intersection of centre ray lies within valid interval of t_hit and wether all the rays intersect the surface
  if( rtPotentialIntersection( (float)t.t_baseRay ) ) 
  {
    // set aperture to infinity for waist rays and divergence rays
    params.apertureType=AT_INFTY;
    t.t_waistRayX=intersectRayPlaneSurface(prd.waistRayX.position, prd.waistRayX.direction, params);
    t.t_waistRayY=intersectRayPlaneSurface(prd.waistRayY.position, prd.waistRayY.direction, params);
    t.t_divRayX=intersectRayPlaneSurface(prd.divRayX.position, prd.divRayX.direction, params);
	t.t_divRayY=intersectRayPlaneSurface(prd.divRayY.position, prd.divRayY.direction, params);
	/**************************************************************************************************************/
	/* what should we do if one of the rays doesn't hit the geometry, the centre ray hit ?                        */
	/* So far we call the hit functions anyway and terminate the ray with an error in the closest hit function    */
	/**************************************************************************************************************/
	
	// calc the geometric normals of all the gaussian beam rays
	geometric_normal=calcHitParams(t);
	// communicate t_hit to closest_hit function
	t_hit=t;
	// pass geometryID to hit-program
	geometryID=params.geometryID;
	// call any hit function with material indexed zero
	rtReportIntersection(0);
  }

}

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(boxmin, boxmax);
}
