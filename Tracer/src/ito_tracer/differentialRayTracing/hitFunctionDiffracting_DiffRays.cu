#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include <optix.h>
#include <optix_math.h>
#include "../rayData.h"
#include "../rayTracingMath.h"
#include "MaterialRefracting_DiffRays_hit.h"

/****************************************************************************/
/*				variable definitions										*/
/****************************************************************************/

rtDeclareVariable(Mat_DiffRays_hitParams, hitParams, attribute hitParams, ); 
rtDeclareVariable(double, t_hit, attribute t_hit, ); 
rtDeclareVariable(MatRefracting_DiffRays_params, params, , ); 
rtDeclareVariable(int,               max_depth, , );
rtDeclareVariable(float,               min_flux, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float,             scene_epsilon, , );
rtDeclareVariable(diffRayStruct, prd, rtPayload, );
//rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(int,               geometryID, attribute geometryID , );

/****************************************************************************/
/*				device functions											*/
/****************************************************************************/

__forceinline__ __device__ void diffractingDiffRays_anyHit_device()
{
  // this material is opaque, so it fully attenuates all shadow rays
  //prd_shadow.attenuation = make_float3(0);
  //rtTerminateRay();
//  if (prd.currentGeometryID == geometryID)
//  {
//    rtIgnoreIntersection();
//  }

}

__forceinline__ __device__ void diffractingDiffRays_closestHit_device( Mat_DiffRays_hitParams hitParams, double t_hit )
{
    bool coat_reflected =false;
    hitRefracting_DiffRays(prd, hitParams, params, t_hit, geometryID, coat_reflected);

  if ( (prd.depth>max_depth) || (prd.flux<min_flux) )
	  prd.running=false;  
}

/********************************************************************************/
/*					OptiX programs												*/
/********************************************************************************/

RT_PROGRAM void anyHit()
{
  diffractingDiffRays_anyHit_device();
}


RT_PROGRAM void closestHit()
{
  diffractingDiffRays_closestHit_device( hitParams, t_hit );
}
