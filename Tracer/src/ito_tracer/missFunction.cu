#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include <optix.h>
#include <optix_math.h>
#include "rayData.h"

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(rayStruct, prd, rtPayload, ); // get per-ray-data structure

RT_PROGRAM void miss()
{
//	rtPrintf("ray stopped in miss!! \n");
	prd.running=false; // stop ray
  //prd_position.position = bg_color;
	//prd_position.position.x=1.0f;
	//prd_position.position.y=2.0f;
	//prd_position.position.z=3.33f;

  //prd.position=make_double3((double)100);
}
