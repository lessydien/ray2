#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>
#include "rayData.h"
#include "CylLenseSurface_Intersect.h"

rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(rayStruct, prd, rtPayload, ); // get per-ray-data structure
rtDeclareVariable(simMode, mode, , );
rtDeclareVariable(CylLenseSurface_ReducedParams, params, , ); // normal vector to surface. i.e. part of the definition of the plane surface geometry
rtDeclareVariable(int, materialListLength, , ); 
// variables that are communicate to the hit program via the attribute mechanism
rtDeclareVariable(Mat_hitParams, hitParams, attribute hitParams, ); // normal to the geometry at the hit-point. at a plane surface this will simply be the normal of the definition of the plane surface
rtDeclareVariable(int,               geometryID, attribute geometryID , );
rtDeclareVariable(double, t_hit, attribute t_hit, ); 

/* calc normal to surface at intersection point */
__forceinline__ __device__ Mat_hitParams calcHitParams(double t)
{
  return calcHitParamsCylLenseSurface(prd.position+t*prd.direction, params);
}

/* calc intersection of ray with geometry */
RT_PROGRAM void intersect(int)
{
  double t;
  // we only calculate the intersection in nonsequential mode or if the current geometry is the next to intersect in "sequential mode"
//  if ( (mode==SIM_GEOMRAYS_NONSEQ) || (prd.currentGeometryID==params.geometryID-1) )
//  {
	  t = intersectRayCylLenseSurface(prd.position, prd.direction, params);

	  // check wether intersection lies within valid interval of t_hit
	  if( rtPotentialIntersection( (float)t ) ) 
	  {
		hitParams=calcHitParams(t);
		// communicate t_hit to closest_hit function
		t_hit=t;
		// pass geometryID to hit-program
		geometryID=params.geometryID;
		// call any hit function of the respective material
		rtReportIntersection( 0 );
//	  }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(boxmin, boxmax);
}
