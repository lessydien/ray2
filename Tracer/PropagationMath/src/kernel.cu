#include "hip/hip_runtime.h"
/***********************************************************************
 This file is part of ITO-MacroSim.

    ITO-MacroSim is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ITO-MacroSim is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Foobar.  If not, see <http://www.gnu.org/licenses/>.
************************************************************************/

#include "kernel.h"
#include "math.h"

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMul(hipDoubleComplex* a, const hipDoubleComplex* b, int size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
        a[i] = hipCmul(a[i], b[i]);     
} 

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulandScale(hipDoubleComplex* a, const hipDoubleComplex* b, int size, double scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
	{
        a[i] = hipCmul(a[i], b[i]);     
		a[i].x = a[i].x*scale;
		a[i].y = a[i].y*scale;
	}
} 

//__global__ void kernel(int *a, int*b)
//{
//	int tx = threadIdx.x;
//	
//	switch(tx)
//	{
//		case 0:
//			*a=*a+10;
//			break;
//		case 1:
//			*b=*b+3;
//			break;
//		default:
//			break;
//	}
//}

static __global__ void scalar_RichardsonWolf_kernel(hipDoubleComplex* Uin_ptr, double* x1_ptr,  double* y1_ptr, double* x2_ptr, double*y2_ptr, unsigned int dimx, unsigned int dimy, unsigned int TileWidth, unsigned int TileHeight, double wvl, double f, double Dz)
{
	unsigned int jx=blockIdx.x*TileWidth+threadIdx.x;
	unsigned int jy=blockIdx.y*TileHeight+threadIdx.y;

	double dx1=abs(x1_ptr[1]-x1_ptr[0]);
	double dy1=abs(y1_ptr[1]-y1_ptr[0]);
	x2_ptr[jx]=(-1.0*dimx/2+jx)/(dimx*dx1)*wvl*f;
	y2_ptr[jy]=(-1.0*dimy/2+jy)/(dimy*dy1)*wvl*f;

	double sigmaX=-x1_ptr[jx]/f;
	double sigmaY=-y1_ptr[jy]/f;

	double GktSqr=1-sigmaX*sigmaX-sigmaY*sigmaY;
	// free space propagation filters out evanescent waves...
	if (GktSqr<0)
	{
		GktSqr=0.0;
		Uin_ptr[jx+jy*dimy]=make_hipDoubleComplex(0.0,0.0);
	}
	else
	{
		// this looks kind of ugly because cudas complex<double> implementation doesn't have any operator notation...
		//Uin_ptr[jx+jy*dimy]=make_hipDoubleComplex(0.0,-1.0)*f*Uin_ptr[jx+jy*dimy]/pow(make_hipDoubleComplex(1-sigmaX*sigmaX-sigmaY*sigmaY,0.0),0.25)*make_hipDoubleComplex(cos(2*PI/wvl*Dz*sqrt(GktSqr)),sin(2*PI/wvl*Dz*sqrt(GktSqr)));
		hipDoubleComplex help=hipCmul(make_hipDoubleComplex(f/pow(1-sigmaX*sigmaX-sigmaY*sigmaY,0.25),0.0),Uin_ptr[jx+jy*dimy]);
		help=hipCmul(make_hipDoubleComplex(0.0,-1.0),help);
		Uin_ptr[jx+jy*dimy]=hipCmul(help,make_hipDoubleComplex(cos(2*PI/wvl*Dz*sqrt(GktSqr)),sin(2*PI/wvl*Dz*sqrt(GktSqr))));
	}
}

bool scalar_RichardsonWolf_wrapper(complex<double>* Uin_ptr, unsigned int dimx, unsigned int dimy, double wvl, double* x1_ptr, double* y1_ptr, double f, double Dz, double** x2_ptrptr, double** y2_ptrptr)
{
	// we handle only regularly squared grids here
	if (dimx!=dimy)
		return 0;

	double k=2*PI/wvl;
	double dx1=abs(x1_ptr[0]-x1_ptr[1]);
	double dy1=abs(y1_ptr[0]-y1_ptr[1]);
	if (dx1!=dy1)
		return 0;

	// transfer data to GPU
	double* x2_kernel_ptr;
	cutilSafeCall(hipMalloc((void**)&x2_kernel_ptr, sizeof(double)*dimx));
	//(hipMalloc((void**)&x2_kernel_ptr, sizeof(double)*dimx));
	double* y2_kernel_ptr;
	cutilSafeCall(hipMalloc((void**)&y2_kernel_ptr, sizeof(double)*dimy));
	//(hipMalloc((void**)&y2_kernel_ptr, sizeof(double)*dimy));

	double* x1_kernel_ptr;
	cutilSafeCall(hipMalloc((void**)&x1_kernel_ptr, sizeof(double)*dimx));
	//(hipMalloc((void**)&x1_kernel_ptr, sizeof(double)*dimx));
	cutilSafeCall(hipMemcpy(x1_kernel_ptr, x1_ptr, sizeof(double)*dimx, hipMemcpyHostToDevice));
	//(hipMemcpy(x1_kernel_ptr, x1_ptr, sizeof(double)*dimx, hipMemcpyHostToDevice));
	double* y1_kernel_ptr;
	cutilSafeCall(hipMalloc((void**)&y1_kernel_ptr, sizeof(double)*dimy));
	//(hipMalloc((void**)&y1_kernel_ptr, sizeof(double)*dimy));
	cutilSafeCall(hipMemcpy(y1_kernel_ptr, y1_ptr, sizeof(double)*dimy, hipMemcpyHostToDevice));
	//(hipMemcpy(y1_kernel_ptr, y1_ptr, sizeof(double)*dimy, hipMemcpyHostToDevice));

	complex<double>* Uin_kernel_ptr;
	cutilSafeCall(hipMalloc((void**)&Uin_kernel_ptr, sizeof(complex<double>)*dimx*dimy));
	//(hipMalloc((void**)&Uin_kernel_ptr, sizeof(complex<double>)*dimx*dimy));
	cutilSafeCall(hipMemcpy(Uin_kernel_ptr, Uin_ptr, sizeof(complex<double>)*dimx*dimy, hipMemcpyHostToDevice));
	//(hipMemcpy(Uin_kernel_ptr, Uin_ptr, sizeof(complex<double>)*dimx*dimy, hipMemcpyHostToDevice));

	unsigned int tileWidth=16;
	unsigned int tileHeight=16;

	dim3 dimBlock(tileWidth,tileHeight,1); // number of threads within each block in x,y,z (maximum of 512 in total. I.e. 512,1,1 or 8,16,2 or ...
	dim3 dimGrid(dimx/tileWidth,dimy/tileHeight,1); // number of blocks in x,y,z (maximum of 65535 for each dimension)

	scalar_RichardsonWolf_kernel<<<dimGrid, dimBlock>>>((hipDoubleComplex*)Uin_kernel_ptr, x1_kernel_ptr, y1_kernel_ptr, x2_kernel_ptr, y2_kernel_ptr, dimx, dimy, tileWidth, tileHeight, wvl, f, Dz);


	// allocate host memory for observation plane coordinates
	double *x2_l=(double*)calloc(dimx,sizeof(double));
	double *y2_l=(double*)calloc(dimy,sizeof(double));

	// transfer coordinates from GPU
	cutilSafeCall(hipMemcpy(x2_l, x2_kernel_ptr, sizeof(double)*dimx, hipMemcpyDeviceToHost));
	//(hipMemcpy(x2_l, x2_kernel_ptr, sizeof(double)*dimx, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(y2_l, y2_kernel_ptr, sizeof(double)*dimy, hipMemcpyDeviceToHost));
	//(hipMemcpy(y2_l, y2_kernel_ptr, sizeof(double)*dimy, hipMemcpyDeviceToHost));

	//deallocate coordinates on GPU
	hipFree(x1_kernel_ptr);
	hipFree(x2_kernel_ptr);
	hipFree(y1_kernel_ptr);
	hipFree(y2_kernel_ptr);

	// do fft
    // plan fft
    hipfftHandle plan;
    //cufftSafeCall(hipfftPlan2d(&plan,dimx, dimy, HIPFFT_Z2Z));
	(hipfftPlan2d(&plan,dimx, dimy, HIPFFT_Z2Z));

    // execute fft
    cufftSafeCall(hipfftExecZ2Z(plan, (hipfftDoubleComplex *)Uin_kernel_ptr, (hipfftDoubleComplex *)Uin_kernel_ptr, HIPFFT_FORWARD));
	//(hipfftExecZ2Z(plan, (hipfftDoubleComplex *)Uin_kernel_ptr, (hipfftDoubleComplex *)Uin_kernel_ptr, HIPFFT_FORWARD));

	// transfer optical field from GPU
	cutilSafeCall(hipMemcpy(Uin_ptr, Uin_kernel_ptr, sizeof(complex<double>)*dimy*dimx, hipMemcpyDeviceToHost));
	//(hipMemcpy(Uin_ptr, Uin_kernel_ptr, sizeof(complex<double>)*dimy*dimx, hipMemcpyDeviceToHost));
	// deallocate optical field on GPU
	hipFree(Uin_kernel_ptr);
	// destroy fft plan
	hipfftDestroy(plan);


	// return pointer to new coordinates
	*x2_ptrptr=x2_l;
	*y2_ptrptr=y2_l;

	return 1;
}